#include "hip/hip_runtime.h"
#include "materials/dielectric.hpp"

#include "hit_details.hpp"
#include "utils.hpp"
#include "random_num_gen.hpp"

namespace material
{
    __device__ maybe_ray dielectric::scatter_ray(const math::ray_t& ray, const hit_details_t &hit_details) const
    {
        // The image being used for this derivation can be found in slide 13 of this ppt : 
        // https://web.cse.ohio-state.edu/~shen.94/681/Site/Slides_files/reflection_refraction.pdf.

        // The refracted ray is T, and the known values are N, V, theta i, theta t.
        // T is decomposed into sum of parallel and perpendicular components.
        // i.e, T = a + b
        // T = M sin(theta t) - N cos(theta t)
        // Now, M is the unknown in this equation.
        // See the above half (The half with V, N, theta i, etc.
        // Here, decomposing V into parallel and perpendicular components, we have:
        // V = N cos(theta i) + (-M)sin(theta i)
        // M = (N cos(theta i) - v) / sin(theta i)
        // Substituting in the equation involving T,
        // T = sin(theta t) (N cos (theta i) - v) / sin(theta i) - N cos(theta t)
        // Now, using snells law, sin (theta i) ei = sin (theta t) et.
        // We do not know sin (theta t), so substitute that with sin (theta i) ei / et.
        // T = (ei / et) (N cos (theta i) - v) - N * (sqrt(1 - sin(theta t)*2)
        // T = (ei / et) (N cos (theta i) - v) - N * sqrt(1  - sin(theta i) * sin(theta i) * e * e) 
        // T = (ei / et) (N cos (theta i) - v) - (N ) * sqrt(1.0f - sin(theta i) * sin(theta i) * e * e)
        // Here, ei / et is basically ior (e)

        float refraction_ratio = hit_details.back_face ? ior : 1.0f / ior;

        const auto& N = hit_details.normal;
        const auto& V = -ray.direction.normalize();

        const auto cos_theta_i = min(math::float3::dot(V, N), 1.0f);
        const auto sin_theta_i = sqrt(1 - cos_theta_i * cos_theta_i);

        // Before proceeding, note that the formula for computation of T requires a sqrt(1 - (sin(theta_i) * e) ^ 2).
        // The terms inside the sqrt cannot be negative. We can use this condition to determine when ray should refract (i.e when the sqrt computation is possible)
        // and when to reflect (when this computation is impossible).

        // Reflect if the below condition is true.
        // Also, we will reflect in some cases where the angle between V and N is large (i.e in such grazing angles
        // any surface will start to exhibit some mirror like reflection properties.
        // We can use the schlick approximation for this.
        const auto schlick_approximation = [&](const auto cos_theta, const auto ior)
        {
            const auto r0 = (1.0f - ior) / (1.0f + ior);
            const auto r0_square = r0 * r0;

            return r0_square + (1 - r0_square) *  std::pow((1.0f - cos_theta), 5);
        };

        // Perform reflection if either refraction is not possible, or if reflectance is above a random value.
        if (refraction_ratio * sin_theta_i > 1.0f || schlick_approximation(cos_theta_i, refraction_ratio) > get_random_float_in_range_0_1())
        {
           return maybe_ray{hit_details.point_of_intersection, (ray.direction - N * 2.0f * math::float3::dot(N, ray.direction)).normalize()};
        }

        // Refract the ray if refraction is possible. 
        const auto T = (N * cos_theta_i  - V) * refraction_ratio - N * std::sqrt(1.0f - sin_theta_i * sin_theta_i * refraction_ratio * refraction_ratio); 

        return maybe_ray(hit_details.point_of_intersection, T.normalize());   
    }
}
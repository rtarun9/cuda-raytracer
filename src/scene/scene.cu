#include "hip/hip_runtime.h"
#include "scene/scene.hpp"

#include "scene/sphere.hpp"
#include "math/ray.hpp"

#include "hit_details.hpp"
#include "utils.hpp"

namespace scene
{
    __host__ scene_t::scene_t(const u32 max_sphere_count, const u32 max_material_count)
        : max_sphere_count(max_sphere_count), max_material_count(max_material_count)
    {
        utils::cuda_check(hipMallocManaged(&spheres, sizeof(sphere_t) * max_sphere_count));
        utils::cuda_check(hipMallocManaged(&materials, sizeof(material::material_t*) * max_material_count));
    }

    scene_t::~scene_t()
    {
        for (int i = 0; i < num_materials; i++)
        {
            hipFree(materials[i]);
        }

        hipFree(materials);
        hipFree(spheres);
    }

    void scene_t::add_sphere(sphere_t& sphere)
    {
        if (num_spheres == max_sphere_count)
        {
            std::cout << "Not adding sphere to scene due to exceeding max sphere count.\n";
            return;
        }

        spheres[num_spheres++] = sphere;
        std::cout << "Added sphere with center : " << spheres[num_spheres - 1].center;
        std::cout << num_spheres << std::endl;
    }
        
    uint32_t scene_t::add_material(material::material_t* mat)
    {
        if (num_materials == max_material_count)
        {
            std::cout << "Not adding material to scene due to exceeding max material count.\n";
            return 0; 
        }


        materials[num_materials++] = (mat);
        std::cout << *(materials + num_materials - 1) << std::endl;
        return num_materials - 1;
    }

    __device__ hit_details_t scene_t::ray_hit(const math::ray_t& ray) const
    {
        // The value of t is slightly greater than 0 because of shadow acne.
        // There are situations where due to floating point precision problems, we may have 
        // a case where the ray param at t is not the exact value, causing the intersection point to be 
        // beneath the surface rather than on it. This will cause the ray to continuously intersect the surface of the sphere,
        // causing continous intersections results in false - shadowing.
        // By setting min_t to be greater than 0, this problem can be resolved.
        float min_t = 0.01f;
        float max_t = 3.40282347e+8F;

        hit_details_t ray_hit_details{};
        ray_hit_details.ray_param_t = -1.0f;
        
        bool ray_hit_object_in_scene = false;

        for (int i = 0; i < num_spheres; i++)
        {
            const sphere_t& sphere = spheres[i];
            const auto t = sphere.hit_by_ray(ray, min_t, max_t);
            if (t != -1.0f)
            {
                // Fill hit_details struct.
                ray_hit_details.ray_param_t = t;
                ray_hit_details.point_of_intersection = ray.at(t);

                math::float3 normal = (ray_hit_details.point_of_intersection - sphere.center) / sphere.radius;

                // To find if the ray hit a back face or front face.
                // If the angle between normal and ray direction is greater than 90, then
                // the ray hit a front face.
                const auto normal_and_ray_direction_dot_product = math::float3::dot(ray.direction.normalize(), normal);
                if (normal_and_ray_direction_dot_product < 0.0f)
                {
                    ray_hit_details.back_face = false;
                    ray_hit_details.normal = normal;
                }
                else
                {
                    ray_hit_details.back_face = true;
                    ray_hit_details.normal = normal * -1.0f;
                }

                ray_hit_details.material_index =sphere.mat_index; 

                max_t = t;

                ray_hit_object_in_scene = true;
            }
        }

        if (!ray_hit_object_in_scene)
        {
            ray_hit_details.ray_param_t = -1.0f;
        } 

        return ray_hit_details;
    }
}
